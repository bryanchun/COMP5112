#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <algorithm>
#include <chrono>

using namespace std;

#include "cuda_smith_waterman.h"

/*
 *  You can add helper functions and variables as you wish.
 */

const int h_tileHeight = 100;
__const__ int d_tileHeight = h_tileHeight;

const int h_MAX_SEQ_SIZE = 20001;
__const__ int d_MAX_SEQ_SIZE = h_MAX_SEQ_SIZE;

__device__ int d_max_score;

// +2 for offsetting tile's top 2 rows
inline __device__
int lin_idx(int x, int y) {
	return utils::dev_idx(x+2, y, d_MAX_SEQ_SIZE);
}

__global__
void kernel(
	char* d_a /* in */, char* d_b /* in */, int a_len /* in */,
	int* d_score /* out */,
	int d /* in; diagonal idx */, int w /* in; width of this diagonal */
) {
	// Compute one or more element on a diagonal 'd'
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int d_tiled = d % d_tileHeight;

	// If there are too many threads, only a subset of threads within 'w' will work
	if (i < w) {
		int numThreads = gridDim.x * blockDim.x,
			numFullCycles = w / numThreads,
			numOutstandingCycles = w % numThreads;
		int numCycles = (numFullCycles == 0) ?
			// Too many threads, thread 'i' will work only once
			1 :
			// If there are too few threads, thread 'i' will work multiple times cyclically by coalesced access
			(numFullCycles + (i < numOutstandingCycles));

		// printf("> (w, d, i) = (%d, %d, %d) has (numThreads, numCycles) = (%d, %d)\n", w, d, i, numThreads, numCycles);
		for (int cycle = 0; cycle < numCycles; cycle++, i += numThreads) {

			// Map from (d, i) -> (x, y) for 'd_a' and 'd_b'
			int delta = max(d - a_len + 1, 0);
			int y = i + delta;
			int x = d - y;

			int current = lin_idx(d_tiled, i);
			/*
			1. d < a_len  -> only pad for lastlast, lastL outside of leftmost bound; left-parallelogram
			2. d == a_len -> no pad; forward triangle
			3. d > a_len  -> no pad; right-parallelogram
			*/
			int lastlast, lastL, lastR;
			if (d < a_len) {
				lastlast = (i == 0) ? 0 : d_score[lin_idx(d_tiled-2, i-1)];
				lastL 	 = (i == 0) ? 0 : d_score[lin_idx(d_tiled-1, i-1)];
				lastR 	 = d_score[lin_idx(d_tiled-1, i)];
			} else {
				lastlast = d_score[lin_idx(d_tiled-2, ((d == a_len) ? i : i+1))];
				lastL 	 = d_score[lin_idx(d_tiled-1, i)];
				lastR	 = d_score[lin_idx(d_tiled-1, i+1)];
				//printf("* (d, i, x, y) = (%d, %d, %d, %d): L of (%d, %d), l of (%d, %d), r of (%d, %d)\n", d, i, x, y, d-2, ((d == a_len) ? i : i+1), d-1, i, d-1, i+1);
			}
			
			d_score[current] =  max(0,
						max(lastlast + sub_mat(d_a[x], d_b[y]),
						max(lastL - GAP,
							lastR - GAP
						)));
			atomicMax(&d_max_score, d_score[current]);
			// printf("> (d, d_tiled, i, x, y) = (%d, %d, %d, %d, %d)\n\t(cIdx, L, l, r, d_a[x], d_b[y]) -> (%d, %d, %d, %d, %c, %c)\n\t(score, d_max_score) = (%d, %d)\n", d, d_tiled, i, x, y, current, lastlast, lastL, lastR, d_a[x], d_b[y], d_score[current], d_max_score);
		}
	}
}

__global__
void nextTile(int* d_score, int d_score_height, int d_score_width) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	// If there are too many threads, only a subset of threads within 'd_score_width' will work
	if (i < d_score_width) {
		int numThreads = gridDim.x * blockDim.x,
			numFullCycles = d_score_width / numThreads,
			numOutstandingCycles = d_score_width % numThreads;
		int numCycles = (numFullCycles == 0) ?
			// Too many threads, thread 'i' will work only once
			1 :
			// If there are too few threads, thread 'i' will work multiple times cyclically by coalesced access
			(numFullCycles + (i < numOutstandingCycles));
		for (int cycle = 0; cycle < numCycles; cycle++, i += numThreads) {
			d_score[lin_idx(-2, i)] = d_score[lin_idx(d_score_height-2, i)];
			d_score[lin_idx(-1, i)] = d_score[lin_idx(d_score_height-1, i)];
			for (int y = 0; y < d_score_height; y++) {
				d_score[lin_idx(y, i)] = 0;
			}
		}
	}
}

__global__
void debug_xy(char* d_a, char* d_b, int a_len, int b_len, int* d_score) {
	printf("\t");
	for (int i = 0; i < b_len; i++) {
        printf("%c\t", d_b[i]);
	}
	printf("\n");

	for (int i = 1; i <= a_len; i++) {
		printf("%c\t", d_a[i-1]);
		for (int j = 1; j <= b_len; j++) {
			int d = (i-1) + (j-1);
			int i_prime = (j-1) - max(d - a_len + 1, 0);
			printf("%d\t", d_score[lin_idx(d, i_prime)]);
		}
		printf("\n");
	}
	printf("\n");
}

__global__
void debug_di(int* d_score, int d_score_height, int d_score_width) {
	for (int d = 0; d < d_score_height; d++) {
		for (int i = 0; i < d_score_width; i++) {
			printf("%d\t", d_score[lin_idx(d, i)]);
		}
		printf("\n");
	}
	printf("\n");
}

int smith_waterman(int blocks_per_grid, int threads_per_block, char *a, char *b, int a_len, int b_len) {
	/*
	 *  Please fill in your codes here.
	 */
	
	// Number of diagonals, Double-max width of a diagonal
	int D = a_len + b_len - 1;
	int max_len = max(a_len, b_len);
	int min_len = min(a_len, b_len);

	// Allocate device memory
	char *d_a, *d_b;
	// Allocate device-global score 'linear matrix'
	int *d_score;
	hipMalloc(&d_a, sizeof(char) * a_len);
	hipMalloc(&d_b, sizeof(char) * b_len);
	// The logical representation is a clipped height at 'h_tileHeight'
	int d_score_height = min(D, h_tileHeight);
	// The actual representation is +2 to include top two rows for offset
	int h_score_height = d_score_height + 2;
	hipMalloc(&d_score, sizeof(int) * (h_score_height * h_MAX_SEQ_SIZE));
	
	// Copy hostToDevice
	hipMemcpy(d_a, a, sizeof(char) * a_len, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, sizeof(char) * b_len, hipMemcpyHostToDevice);
	// Initialise score 'linear matrix' to zeroes
	hipMemset(d_score, 0, sizeof(int) * (h_score_height * h_MAX_SEQ_SIZE));
	// Initialise max_score to zero
	hipMemset(&d_max_score, 0, sizeof(int));
	
	// Invoke kernel
	for (int d = 0; d < D; d++) {
		int w = (d < min_len) ? (d + 1) : ((d < max_len) ? min_len : (D - d));
		//printf("(d, w) = (%d, %d)\n", d, w);
		kernel<<<blocks_per_grid, threads_per_block>>>(d_a, d_b, a_len, d_score, d, w);
		if ((d+1) % h_tileHeight == 0 && d < D-1) {
			// debug_xy<<<1, 1>>>(d_a, d_b, a_len, b_len, d_score);
			// debug_di<<<1, 1>>>(d_score, h_score_height, min_len);
			nextTile<<<blocks_per_grid, threads_per_block>>>(d_score, d_score_height, min_len);
		}
	}
	// debug_xy<<<1, 1>>>(d_a, d_b, a_len, b_len, d_score);
	// debug_di<<<1, 1>>>(d_score, h_score_height, min_len);
	
	// Return h_answer
	int h_answer;
	hipMemcpyFromSymbol(&h_answer, HIP_SYMBOL(d_max_score), sizeof(int), 0, hipMemcpyDeviceToHost);
	
	// Free device memory
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_score);
	
	return h_answer;
}
